#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    int numElements = 50000000;  // Increased for better timing
    size_t size = numElements * sizeof(float);

    // Timing variables
    hipEvent_t start, stop, total_start, total_stop;
    float milliseconds = 0, total_milliseconds = 0;
    auto cpu_start = std::chrono::high_resolution_clock::now();

    // Create CUDA events for total time
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);
    
    // Start total time measurement
    hipEventRecord(total_start);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    auto cpu_end = std::chrono::high_resolution_clock::now();
    auto cpu_duration = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start);
    printf("CPU Initialization: %ld ms\n", cpu_duration.count());

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    hipEventRecord(start);
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Memory Allocation: %.2f ms\n", milliseconds);

    // Copy host arrays to device
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Data Transfer (Host to Device): %.2f ms\n", milliseconds);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution: %.2f ms\n", milliseconds);

    // Copy result back to host
    hipEventRecord(start);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Data Transfer (Device to Host): %.2f ms\n", milliseconds);

    // Stop total time measurement
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_milliseconds, total_start, total_stop);
    printf("Total GPU Time: %.2f ms\n", total_milliseconds);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);

    return 0;
}